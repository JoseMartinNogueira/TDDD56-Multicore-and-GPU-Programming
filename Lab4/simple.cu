#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>

#include <math.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *n, float *c) 
{
	c[threadIdx.x] = sqrt(n[threadIdx.x]);
}

/*__global__ 
void simple(float *c) 
{
	c[threadIdx.x] = threadIdx.x;
}*/

int main()
{
	float *c = new float[N];	
	float *cd;
	const int size = N*sizeof(float);
	float *src = new float[N];
	float *dst;

	for(int i=0; i<N; i++)
		src[i]=i*i*i*i;

	hipMalloc( (void**)&cd, size );	
	hipMalloc( (void**)&dst, size );
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hipMemcpy( dst, src, size, hipMemcpyHostToDevice );
	simple<<<dimGrid, dimBlock>>>(dst,cd);
	//simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	hipFree( dst);
	
	for (int i = 0; i < N; i++)
		printf("%f\n ", c[i]);
	printf("\n");
	delete[] c;
	delete[] src;
	printf("done\n");
	return EXIT_SUCCESS;
}
